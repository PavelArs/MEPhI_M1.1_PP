#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void cpv__(const float *vec, float *res, int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    res[i] = int(vec[i] > 0);
}

extern "C" {
    float *gpu_cpv(float *vec, size_t N) {
        hipError_t err = hipSuccess;
        float *dev_vec, *dev_res, *res;
        int threadsPerBlock = 256;
        int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
        res = new float[blocksPerGrid];

        err = hipMalloc((void **) &dev_vec, N * sizeof(float));
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device vector (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        err = hipMalloc((void **) &dev_res, blocksPerGrid * sizeof(float));
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device vector (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        err = hipMemcpy(dev_vec, vec, N * sizeof(float), hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        cpv__<<<blocksPerGrid, threadsPerBlock>>>(dev_vec, dev_res, N);
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch cpv kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        err = hipMemcpy(res, dev_res, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector from device to host (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        err = hipFree(dev_vec);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to free device vector (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
//        hipFree(dev_res);

        err = hipDeviceReset();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        return res;
    }
}